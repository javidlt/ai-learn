/** Explanation threads 
 * This CUDA program demonstrates a basic kernel that adds two vectors using only thread indices.
 * Each thread computes the sum of corresponding elements from two input arrays and stores the result in a third array.
 * The kernel is launched with a single block containing multiple threads, where each thread processes one element of the arrays.
 * This version reads vector size from user input, generates random vectors on GPU, and measures execution times.
 * Note: This implementation is limited to a maximum of 1024 elements due to the single block configuration.
*/

#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <time.h>

__global__ void setupRandomStates(hiprandState *state, unsigned long seed, int n) {
    int index = threadIdx.x;
    if (index < n) {
        hiprand_init(seed, index, 0, &state[index]);
    }
}

__global__ void generateVector(int *vec, hiprandState *state, int n) {
    int index = threadIdx.x;
    if (index < n) {
        vec[index] = hiprand(&state[index]) % 100; // Random value between 0 and 99
    }
}

__global__ void sumVectors(int *a, int *b, int *c, int n) {
    int index = threadIdx.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int arraySize;
    
    // Read vector size from user
    printf("Enter the vector size: ");
    scanf("%d", &arraySize);
    
    if (arraySize <= 0 || arraySize > 1024) {
        printf("Vector size must be between 1 and 1024 (CUDA block limit)\n");
        return 1;
    }
    
    // Allocate host memory for result
    int *c = (int*)malloc(arraySize * sizeof(int));
    
    int *d_a, *d_b, *d_res;
    hiprandState *d_state;
    size_t size = arraySize * sizeof(int);
    
    // Allocate device memory
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_res, size);
    hipMalloc((void**)&d_state, arraySize * sizeof(hiprandState));
    
    // Setup random states
    setupRandomStates<<<1, arraySize>>>(d_state, time(NULL), arraySize);
    hipDeviceSynchronize();
    
    // Create CUDA events for timing
    hipEvent_t start_gen, stop_gen, start_sum, stop_sum;
    hipEventCreate(&start_gen);
    hipEventCreate(&stop_gen);
    hipEventCreate(&start_sum);
    hipEventCreate(&stop_sum);
    
    // Time vector generation
    printf("\nGenerating random vectors...\n");
    hipEventRecord(start_gen);
    
    generateVector<<<1, arraySize>>>(d_a, d_state, arraySize);
    generateVector<<<1, arraySize>>>(d_b, d_state, arraySize);
    
    hipEventRecord(stop_gen);
    hipEventSynchronize(stop_gen);
    
    // Calculate generation time
    float genTime;
    hipEventElapsedTime(&genTime, start_gen, stop_gen);
    
    // Time vector summation
    printf("Computing vector sum...\n");
    hipEventRecord(start_sum);
    
    sumVectors<<<1, arraySize>>>(d_a, d_b, d_res, arraySize);
    
    hipEventRecord(stop_sum);
    hipEventSynchronize(stop_sum);
    
    // Calculate summation time
    float sumTime;
    hipEventElapsedTime(&sumTime, start_sum, stop_sum);
    
    // Copy result array from device to host
    hipMemcpy(c, d_res, size, hipMemcpyDeviceToHost);
    
    // Print timing results
    printf("\n=== Timing Results ===\n");
    printf("Vector size: %d elements\n", arraySize);
    printf("Vector generation time: %.6f ms\n", genTime);
    printf("Vector summation time: %.6f ms\n", sumTime);
    printf("Total computation time: %.6f ms\n", genTime + sumTime);
    
    // Clean up
    hipEventDestroy(start_gen);
    hipEventDestroy(stop_gen);
    hipEventDestroy(start_sum);
    hipEventDestroy(stop_sum);
    
    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);
    hipFree(d_state);
    
    // Free host memory
    free(c);
    
    return 0;
}