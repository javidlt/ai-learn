/** Explanation blocks and threads
 * This CUDA program demonstrates a kernel that efficiently uses both blocks and threads to add two vectors.
 * Unlike the thread-only version, this can handle arrays of any size (not limited to 1024 elements).
 * Each thread computes its global index using: blockIdx.x * blockDim.x + threadIdx.x
 * This allows parallel processing across multiple blocks, each containing multiple threads.
 * The grid is automatically calculated based on the array size and block size.
 * We cannot have more than 1024 threads per block, so we define a constant THREADS_PER_BLOCK.
*/

#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define THREADS_PER_BLOCK 256

__global__ void setupRandomStates(hiprandState *state, unsigned long seed, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        hiprand_init(seed, index, 0, &state[index]);
    }
}

__global__ void generateVector(int *vec, hiprandState *state, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        vec[index] = hiprand(&state[index]) % 100; // Random value between 0 and 99
    }
}

__global__ void sumVectors(int *a, int *b, int *c, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int arraySize;

    // Read vector size from user
    printf("Enter the vector size: ");
    scanf("%d", &arraySize);

    if (arraySize <= 0) {
        printf("Vector size must be greater than 0\n");
        return 1;
    }

    // Calculate grid dimensions
    int blocksPerGrid = (arraySize + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    printf("\n=== Configuration ===\n");
    printf("Vector size: %d elements\n", arraySize);
    printf("Threads per block: %d\n", THREADS_PER_BLOCK);
    printf("Blocks per grid: %d\n", blocksPerGrid);
    printf("Total threads launched: %d\n", blocksPerGrid * THREADS_PER_BLOCK);

    // Allocate host memory for result
    int *c = (int*)malloc(arraySize * sizeof(int));

    int *d_a, *d_b, *d_res;
    hiprandState *d_state;
    size_t size = arraySize * sizeof(int);

    // Allocate device memory
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_res, size);
    hipMalloc((void**)&d_state, arraySize * sizeof(hiprandState));

    // Setup random states
    setupRandomStates<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_state, time(NULL), arraySize);
    hipDeviceSynchronize();

    // Create CUDA events for timing
    hipEvent_t start_gen, stop_gen, start_sum, stop_sum;
    hipEventCreate(&start_gen);
    hipEventCreate(&stop_gen);
    hipEventCreate(&start_sum);
    hipEventCreate(&stop_sum);

    // Time vector generation
    printf("\nGenerating random vectors...\n");
    hipEventRecord(start_gen);

    generateVector<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_a, d_state, arraySize);
    generateVector<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_b, d_state, arraySize);

    hipEventRecord(stop_gen);
    hipEventSynchronize(stop_gen);

    // Calculate generation time
    float genTime;
    hipEventElapsedTime(&genTime, start_gen, stop_gen);

    // Time vector summation
    printf("Computing vector sum...\n");
    hipEventRecord(start_sum);

    sumVectors<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_a, d_b, d_res, arraySize);

    hipEventRecord(stop_sum);
    hipEventSynchronize(stop_sum);

    // Calculate summation time
    float sumTime;
    hipEventElapsedTime(&sumTime, start_sum, stop_sum);

    // Copy result array from device to host
    hipMemcpy(c, d_res, size, hipMemcpyDeviceToHost);

    // Print timing results
    printf("\n=== Timing Results ===\n");
    printf("Vector generation time: %.6f ms\n", genTime);
    printf("Vector summation time: %.6f ms\n", sumTime);
    printf("Total computation time: %.6f ms\n", genTime + sumTime);

    // Clean up
    hipEventDestroy(start_gen);
    hipEventDestroy(stop_gen);
    hipEventDestroy(start_sum);
    hipEventDestroy(stop_sum);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);
    hipFree(d_state);

    // Free host memory
    free(c);

    return 0;
}
